#include "hip/hip_runtime.h"
/// @file
////////////////////////////////////////////////////////////////////////////////////////////////////
///
/// Copyright (C) 2016/17      Christian Lessig, Otto-von-Guericke Universitaet Magdeburg
///
////////////////////////////////////////////////////////////////////////////////////////////////////
///
///  module     : lecture 6
///
///  author     : lessig@isg.cs.ovgu.de
///
///  project    : GPU Programming
///
///  description: reduction in Cuda
///
////////////////////////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <iostream>
#include <vector>
#include <chrono>

typedef std::chrono::time_point<std::chrono::high_resolution_clock> tpoint;

// includes, project
#include "cuda_util.h"

////////////////////////////////////////////////////////////////////////////////////////////////////
// initialize Cuda device
////////////////////////////////////////////////////////////////////////////////////////////////////
void
initDevice( int& device_handle, unsigned int& max_threads_per_block) {

  int deviceCount = 0;
  checkErrorsCuda( hipGetDeviceCount(&deviceCount));

  if( 0 == deviceCount) {
    std::cerr << "initDevice() : No CUDA device found." << std::endl;
  }

  // one could implement more complex logic here to find the fastest device
  if( deviceCount > 1) {
    std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
  }

  // set the device
  checkErrorsCuda( hipSetDevice( device_handle));

  hipDeviceProp_t device_props;
  checkErrorsCuda( hipGetDeviceProperties(&device_props, device_handle));
  max_threads_per_block = device_props.maxThreadsPerBlock;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// initialize device memory
////////////////////////////////////////////////////////////////////////////////////////////////////
void
initDeviceMemory( const std::vector<int>& data, int*& data_device, const unsigned int size) {

  // allocate device memory
  checkErrorsCuda( hipMalloc((void **) &data_device, sizeof(int) * size));

  // copy device memory
  checkErrorsCuda( hipMemcpy( data_device, &data[0], sizeof(int) * size, hipMemcpyHostToDevice));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// initialize device memory
////////////////////////////////////////////////////////////////////////////////////////////////////
void
getResultDevice( const int* data_device, std::vector<int>& data, const unsigned int size) {

  checkErrorsCuda(hipMemcpy( &data[0], data_device, sizeof(int) * size, hipMemcpyDeviceToHost));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// free device memory
////////////////////////////////////////////////////////////////////////////////////////////////////
void
freeDeviceMemory( int*& data_device) {

  checkErrorsCuda( hipFree( data_device));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// reduction
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void
reduction( int* data, unsigned int size) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = 1;

  while( stride < size) {
    if( 0 == (tid % (2*stride))) {
      data[tid] = data[tid] + data[tid+stride];
    }
    stride *= 2;
  }

}

////////////////////////////////////////////////////////////////////////////////////////////////////
// program entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
int
main( int /*argc*/, char** /*argv*/ ) {

  // initialize device
  int device_handle = 0;
  unsigned int max_threads_per_block = 0;
  initDevice( device_handle, max_threads_per_block);

  // set up host memory
  // size is chosen so that two reduction steps would suffice
  const unsigned int size = 64 * max_threads_per_block * max_threads_per_block;
  std::vector<int> data( size);
  for( unsigned int i = 0; i < size; ++i) {
    data[i] = 1.0;
  }

  // initialize device memory
  int* data_device = nullptr;
  initDeviceMemory( data, data_device, size);

  // determine thread layout
  int num_threads_per_block = std::min( size, max_threads_per_block);
  int num_blocks = size / max_threads_per_block;
  if( 0 != size % max_threads_per_block) {
    num_blocks++;
  }

  reduction<<< num_blocks , num_threads_per_block >>>( data_device, size);
  checkLastCudaError( "Kernel launch failed.");

  getResultDevice( data_device, data, 1);
  int res = data[0];
  std::cerr << "Result = " << res << std::endl;

  // run again for timing
  hipDeviceSynchronize();
  tpoint t_start = std::chrono::high_resolution_clock::now();
  for( unsigned int k = 0; k < 1024; ++k) {
    reduction<<< num_blocks , num_threads_per_block >>>( data_device, size);
  }
  hipDeviceSynchronize();

  tpoint t_end = std::chrono::high_resolution_clock::now();
  double wall_clock = std::chrono::duration<double, std::milli>(t_end-t_start).count();
  std::cerr << "Execution time: " <<  wall_clock << " ms."<< std::endl;

  checkLastCudaError( "Kernel launch failed.");

  // clean up device memory
  freeDeviceMemory( data_device);

  return EXIT_SUCCESS;
}
