#include "hip/hip_runtime.h"
/// @file
////////////////////////////////////////////////////////////////////////////////////////////////////
///
/// Copyright (C) 2016/17      Christian Lessig, Otto-von-Guericke Universitaet Magdeburg
///
////////////////////////////////////////////////////////////////////////////////////////////////////
///
///  module     : tutorial 5
///
///  author     : lessig@isg.cs.ovgu.de
///
///  project    : GPU Programming
///
///  description: CUDA convolution
///
////////////////////////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <iostream>
#include <algorithm>
#include <chrono>

typedef std::chrono::time_point<std::chrono::high_resolution_clock> tpoint;

// includes, project
#include "cuda_util.h"
#include "kernel_separable.h"
#include "image.h"

// host implementation
extern void
convSeparableHost( float* kdata, const int& kernel_supp, const Image& image, Image& image_conv);

////////////////////////////////////////////////////////////////////////////////////////////////////
// convolution
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void
convSeparable1( float* kernel, const int kernel_supp_half,
                float* image, float* image_conv, const unsigned int image_size) {

    //gridDim - wie viele blöcke gibt es
    int pix_r = thread.Idx.x+blockIdx.x*blockDim.x; //id innerhalb des thread blocks, zwischen 0 und 32=sqrt(2^10)
    int pix_c = threadIdx.y + blockIdx.y * blockDim.y;

    float weight_row = 0.0;
        float weight = 0.0;

        int ik = 0;
        int jk = 0;

        image_conv[pix_r * image_size + col] = 0.0;

        for( int i = pix_r - SuppHalf; i <= pix_r + SuppHalf; ++i, ++ik) {
          weight_row = kernel[ik];
          jk = 0;
          for( int j = pix_c - SuppHalf; j <= pix_c + SuppHalf; ++j, ++jk) {

            if( ( i < 0 || j < 0) || (i >= image_size) || (j >= image_size)) {
              continue;
            }

            weight = weight_row * kernel[jk];
            image_conv[pix_r * image_size + pix_c] += weight * image[i * image_size + j];
          }
        }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// initialize Cuda device
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
initDevice( int& device_handle, int& max_threads_per_block) {

  int deviceCount = 0;
  checkErrorsCuda( hipGetDeviceCount(&deviceCount));

  if( 0 == deviceCount) {
    std::cerr << "initDevice() : No CUDA device found." << std::endl;
    return false;
  }

  // one could implement more complex logic here to find the fastest device
  if( deviceCount > 1) {
    std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
  }

  // set the device
  checkErrorsCuda( hipSetDevice( device_handle));

  hipDeviceProp_t device_props;
  checkErrorsCuda( hipGetDeviceProperties(&device_props, device_handle));
  max_threads_per_block = device_props.maxThreadsPerBlock;

  return true;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// program entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
int
main( int /*argc*/, char** /*argv*/ ) {

  const int kernel_supp = 5;
  const int kernel_supp_half = kernel_supp / 2;
  float kdata[] = {0.0103339f, 0.207561f, 0.56421f, 0.207561f, 0.0103339f};

  Image image( "../images/im.pgm");

  Image image_conv( image.n_rows, image.n_cols);
  convSeparableHost( kdata, kernel_supp_half, image, image_conv);
  image_conv.write( "../images/im_conv_host.pgm");


  // check execution environment
  int device_handle = 0;
  int max_threads_per_block = 0;
  if( ! initDevice( device_handle, max_threads_per_block)) {
    return EXIT_FAILURE;
  }

  // initialize memory
  float* kernel_device = nullptr;
  float* image_device = nullptr;
  float* image_conv_device = nullptr;

  // allocate device memory
  checkErrorsCuda( hipMalloc((void **) &kernel_device, sizeof(float) * kernel_supp));
  checkErrorsCuda( hipMalloc((void **) &image_device, sizeof(float) * image.n_cols * image.n_rows));
  checkErrorsCuda( hipMalloc((void **) &image_conv_device, sizeof(float) * image.n_cols * image.n_rows));

  // copy device memory
  checkErrorsCuda( hipMemcpy( (void*) kernel_device, kdata,
                                sizeof(float) * kernel_supp,
                                hipMemcpyHostToDevice ));
  checkErrorsCuda( hipMemcpy( (void*) image_device, &(image.data[0]),
                                sizeof(float) * image.n_cols * image.n_rows,
                                hipMemcpyHostToDevice ));

  // determine thread layout
  unsigned int sqrt_max_threads = (unsigned int) std::sqrt(max_threads_per_block);
  dim3 num_threads_per_block(sqrt_max_threads,sqrt_max_threads) = num_blocks;
  dim3 num_blocks;
  std::cout << "num_blocks = " << num_blocks.x << " / " << num_blocks.y << std::endl;
  std::cout << "num_threads_per_block = " << num_threads_per_block.x << " / "
                                          << num_threads_per_block.y << std::endl;

  // run kernel
  assert( image.n_rows == image.n_cols);
  tpoint t_start = std::chrono::high_resolution_clock::now();
  convSeparable1<<< num_blocks , num_threads_per_block >>>( kernel_device, kernel_supp_half, image_device,
                                                           image_conv_device, image.n_rows);

  tpoint t_end = std::chrono::high_resolution_clock::now();
  double wall_clock = std::chrono::duration<double, std::milli>(t_end-t_start).count();
  std::cerr << "Execution time: " <<  wall_clock << " ms."<< std::endl;

  checkLastCudaError("Kernel execution failed");
  hipDeviceSynchronize();

  // copy result back to host
  checkErrorsCuda( hipMemcpy( &image_conv.data[0], image_conv_device,
                               sizeof(float) * image.n_cols * image.n_rows,
                               hipMemcpyDeviceToHost ));
  // write result
  image_conv.write( "../images/im_conv_device.pgm");

  // clean up device memory
  checkErrorsCuda( hipFree( kernel_device));
  checkErrorsCuda( hipFree( image_device));
  checkErrorsCuda( hipFree( image_conv_device));

  return EXIT_SUCCESS;
}
